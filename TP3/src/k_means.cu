
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <unistd.h>
#include <omp.h>
#include <string.h>

#define TRUE 1
#define FALSE 0

// Função que calcula a distância entre um ponto e um cluster
__device__ float distanceCalculation(float clusterX, float clusterY, float pointX, float pointY)
{
    return (float)((clusterX - pointX) * (clusterX - pointX)) + ((clusterY - pointY) * (clusterY - pointY));
}

void populate(int *numPoints, float *pointX, float *pointY, int *pointCluster, float *clusterX, float *clusterY, int n, int k, int t)
{
    srand(10);

    for (int i = 0; i < n; i++)
    {
        pointX[i] = (float)rand() / RAND_MAX; // Atribuimos a cada ponto cordenadas random
        pointY[i] = (float)rand() / RAND_MAX; // Atribuimos a cada ponto cordenadas random
        pointCluster[i] = -1;                 // Inicialmente cada amostra ainda não está associada a nenhum cluster
    }

    for (int i = 0; i < k; i++)
    {
        clusterX[i] = pointX[i]; // Seguindo o algoritmo, os primeiros K pontos iniciais são os centroids dos K clusters.
        clusterY[i] = pointY[i];
        numPoints[i]++;      // Atualizamos o numero de pontos associado a cada cluster
        pointCluster[i] = i; // Atribuimos ao ponto o cluster a que está associado
    }
}

// Função para calcular o melhor cluster para cada ponto.
/*
__device__
void bestCluster(float* clusterX, float* clusterY, float pointX, float pointY,  int n, int k, int t){

    int bestCluster = 0;
    float tempDistance = 0;
    float distance = distanceCalculation(clusterX[0], clusterY[0], pointX, pointY);
    for(int i = 1; i < k; i++){
        tempDistance = distanceCalculation(clusterX[i], clusterY[i], pointX, pointY);
        if(tempDistance < distance){
            distance = tempDistance;
            bestCluster = i;
        }
    }
}
*/

__global__ void calculateBestCluster(int *numPoints, float *pointX, float *pointY, int *pointCluster, float *clusterX, float *clusterY, int n, int k, int t)
{

    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride)
    {
        int bClu = -1;
        // bClu = bestCluster(clusterX, clusterY, pointX[i], pointY[i], n, k, t);   // Calcula o melhor cluster para o ponto
        int bestCluster = 0;
        float tempDistance = 0;
        float distance = distanceCalculation(clusterX[0], clusterY[0], pointX[i], pointY[i]);
        for (int j = 1; j < k; j++)
        {
            tempDistance = distanceCalculation(clusterX[j], clusterY[j], pointX[i], pointY[i]);
            if (tempDistance < distance)
            {
                distance = tempDistance;
                bestCluster = i;
            }
        }

        bClu = bestCluster;

        if (bClu != pointCluster[i])
        { // Caso o melhor cluster seja diferente do atribuido anteriormente é atualizado
            if (pointCluster[i] != -1)
                numPoints[pointCluster[i]]--; // Decrementamos o numero de pontos associado a esse cluster

            numPoints[bClu]++; // Incrementamos o numero de pontos associado a esse cluster
        }
        pointCluster[i] = bClu;
    }

    __syncthreads();
}

void calculateCentroid(float *pointX, float *pointY, int *pointCluster, float *clusterX, float *clusterY, int *numPoints, int n, int k, int t)
{

    float xSumCluster[k];
    float ySumCluster[k];

    memset(xSumCluster, 0.0f, k * sizeof(float));
    memset(ySumCluster, 0.0f, k * sizeof(float));

#pragma omp parallel num_threads(t) reduction(+                              \
                                              : xSumCluster[:k]) reduction(+ \
                                                                           : ySumCluster[:k])
    {
#pragma omp for // reduction(+:xSumCluster[:k] ySumCluster[:k])
        for (int i = 0; i < n; i++)
        {
            xSumCluster[pointCluster[i]] += pointX[i];
            ySumCluster[pointCluster[i]] += pointY[i];
        }
    }

    for (int i = 0; i < k; i++)
    {
        clusterX[i] = (float)xSumCluster[i] / numPoints[i];
        clusterY[i] = (float)ySumCluster[i] / numPoints[i];
    }
}

// Função de Print
void printClusters(float *clusterX, float *clusterY, int *numPoints, int n, int k, int t)
{
    printf("N = %d, K = %d\n", n, k);
    for (int i = 0; i < k; i++)
    {
        printf("Center: (%.3f, %.3f) : Size: %d\n", clusterX[i], clusterY[i], numPoints[i]);
    }
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        printf("Cuda error: %s, %s\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}

// Função principal (Algoritmo)
void lloyds(float *pointX, float *pointY, int *pointCluster, float *clusterX, float *clusterY, int n, int k, int t)
{

    int *numPoints = (int *)malloc(k * sizeof(int));
    for (int i = 0; i < k; i++)
        numPoints[i] = 0;

    populate(numPoints, pointX, pointY, pointCluster, clusterX, clusterY, n, k, t);

    int *d_numPoints;
    float *d_pointX;
    float *d_pointY;
    int *d_pointCluster;
    float *d_clusterX;
    float *d_clusterY;
    //    int * d_n;
    //    int * d_k;
    //    int * d_t;

    hipMalloc((void **)&d_numPoints, k * sizeof(int));
    hipMalloc((void **)&d_pointX, n * sizeof(float));
    hipMalloc((void **)&d_pointY, n * sizeof(float));
    hipMalloc((void **)&d_pointCluster, n * sizeof(int));
    hipMalloc((void **)&d_clusterX, k * sizeof(float));
    hipMalloc((void **)&d_clusterY, k * sizeof(float));
    //    cudaMalloc((void**)&d_n, sizeof(int));
    //    cudaMalloc((void**)&d_k, sizeof(int));
    //    cudaMalloc((void**)&d_t, sizeof(int));

    hipMemcpy(d_numPoints, numPoints, k * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_pointX, pointX, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_pointY, pointY, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_pointCluster, pointCluster, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_clusterX, clusterX, k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_clusterY, clusterY, k * sizeof(float), hipMemcpyHostToDevice);
    //  cudaMemcpy(d_n, &n, sizeof(int), cudaMemcpyHostToDevice);
    //  cudaMemcpy(d_k, &k, sizeof(int), cudaMemcpyHostToDevice);
    //  cudaMemcpy(d_t, &t, sizeof(int), cudaMemcpyHostToDevice);

    calculateBestCluster<<<65535, 1024>>>(d_numPoints, d_pointX, d_pointY, d_pointCluster, d_clusterX, d_clusterY, n, k, t);

    hipMemcpy(numPoints, d_numPoints, k * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(pointCluster, d_pointCluster, n * sizeof(int), hipMemcpyDeviceToHost);

    calculateCentroid(pointX, pointY, pointCluster, clusterX, clusterY, numPoints, n, k, t);

    int a = 0;
    while (a < 20)
    {

        hipMemcpy(d_clusterX, clusterX, k * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_clusterY, clusterY, k * sizeof(float), hipMemcpyHostToDevice);
        checkCUDAError("memcpy1");

        calculateBestCluster<<<65535, 1024>>>(d_numPoints, d_pointX, d_pointY, d_pointCluster, d_clusterX, d_clusterY, n, k, t);

        hipMemcpy(numPoints, d_numPoints, k * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(pointCluster, d_pointCluster, n * sizeof(int), hipMemcpyDeviceToHost);
        checkCUDAError("memcpy2");

        calculateCentroid(pointX, pointY, pointCluster, clusterX, clusterY, numPoints, n, k, t);

        a++;
    }

    //   free(d_numPoints);
    //   free(d_pointX);
    //   free(d_pointY);
    //   free(d_pointCluster);
    //   free(d_clusterX);
    //   free(d_clusterY);
    //    free(d_n);
    //    free(d_k);
    //    free(d_t);

    printClusters(clusterX, clusterY, numPoints, n, k, t);
    printf("Iterations: %d\n", a);
    free(numPoints);
}

int main(int argc, char *argv[])
{

    int n;
    int k;
    int t;
    if (argc == 4)
    {
        n = atoi(argv[1]);
        k = atoi(argv[2]);
        t = atoi(argv[3]);
    }
    else
    {
        n = atoi(argv[1]);
        k = atoi(argv[2]);
        t = 1;
    }

    float *pointX = (float *)malloc(n * sizeof(float));
    float *pointY = (float *)malloc(n * sizeof(float));

    int *pointCluster = (int *)malloc(n * sizeof(float));

    float *clusterX = (float *)malloc(k * sizeof(float));
    float *clusterY = (float *)malloc(k * sizeof(float));

    lloyds(pointX, pointY, pointCluster, clusterX, clusterY, n, k, t);
    // tratar de passar a alocação de memoria para o cuda, transferencia de valores.
    free(pointX);
    free(pointY);
    free(pointCluster);
    free(clusterX);
    free(clusterY);
}
